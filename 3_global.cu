#include "hip/hip_runtime.h"

#include "2_equalize.h"

#define INTENSITIES 256

__global__ void apply_function( int* d_blue, int* d_green, int* d_red, 
                                int* d_out_blue, int* d_out_green, int* d_out_red, 
                                int len) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // count 
    if (tid < len) {
        float A = 1.5;
        float B = 30;
        float res_b = (float)d_blue[tid] * A + B;
        float res_g = (float)d_green[tid] * A + B;
        float res_r = (float)d_red[tid] * A + B;
        if (res_b > 255) res_b = 255;
        if (res_g > 255) res_g = 255;
        if (res_r > 255) res_r = 255;

        d_out_blue[tid] = (int)res_b;
        d_out_green[tid] = (int)res_g;
        d_out_red[tid] = (int)res_r;
    }
}

/*
void global_func_cuda(int*** img, int rows, int cols, int channels, int*** o_output1) {
    int* d_img;
    int* d_output1;
    int* zeros = new int[INTENSITIES];
    int length = rows * cols;
    int size = sizeof(int) * length * channels;

    for (int i = 0; i < INTENSITIES; i++)
        zeros[i] = 0.0;

    hipMalloc((void**)&d_img, size);
    hipMalloc((void**)&d_output1, size);
    
    hipMemcpy(d_img, img, size, hipMemcpyHostToDevice);    

    int block_size = rows;
    int grid_size = ((length + block_size) / block_size);
    apply_function << < grid_size, block_size >> > (d_img, d_output1, length);

    hipMemcpy(o_output1, d_output1, sizeof(int) * INTENSITIES, hipMemcpyDeviceToHost);
    
    hipFree(d_img);
    hipFree(d_output1);
}
*/



void global_func_cuda(int* blue, int* green, int* red, int rows, int cols, int *o_blue, int* o_green, int* o_red ) {
    int* d_blue, * d_green, * d_red;
    int* d_out_blue, * d_out_green, * d_out_red;    
    int length = rows * cols; 
    int size = sizeof(int) * length;

    hipMalloc((void**)&d_blue, size);
    hipMalloc((void**)&d_green, size);
    hipMalloc((void**)&d_red, size);
    hipMalloc((void**)&d_out_blue, size);
    hipMalloc((void**)&d_out_green, size);
    hipMalloc((void**)&d_out_red, size);

    hipMemcpy(d_blue, blue, size, hipMemcpyHostToDevice);
    hipMemcpy(d_green, green, size, hipMemcpyHostToDevice);
    hipMemcpy(d_red, red, size, hipMemcpyHostToDevice);

    int block_size = rows;
    int grid_size = ((length + block_size) / block_size);
    apply_function << < grid_size, block_size >> > (d_blue, d_green, d_red, d_out_blue, d_out_green, d_out_red, length);

    hipMemcpy(o_blue, d_out_blue, size, hipMemcpyDeviceToHost);
    hipMemcpy(o_green, d_out_green, size, hipMemcpyDeviceToHost);
    hipMemcpy(o_red, d_out_red, size, hipMemcpyDeviceToHost);

    hipFree(d_blue);
    hipFree(d_green);
    hipFree(d_red);
    hipFree(d_out_blue);
    hipFree(d_out_green);
    hipFree(d_out_red);
}
