#include "hip/hip_runtime.h"

#include "2_equalize.h"

#define INTENSITIES 256

__global__ void hist_frecuencies(int* d_data, int* d_hist, float* d_hist_p, int len) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // count 
    if (tid < len) {
        atomicAdd(&(d_hist_p[d_data[tid]]), 1.0/len);
        atomicAdd(&(d_hist[d_data[tid]]), 1);
    }
}

__global__ void rebuild_img(float* d_CH, int* d_new_img, int* old_img, int len) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // count 
    if (tid < len) {
        d_new_img[tid] = d_CH[old_img[tid]];
    }
}

// esto no se puede paralelizar
void sum_probabilities(float * in, float *out, int len){
    float acc = 0.0;
    for (int i = 0; i < len; i++) {
        acc += in[i];
        out[i] = floor( (INTENSITIES - 1) * acc);
    }
}

void equalize_hist_cuda(int* data, int rows, int cols, int* o_hist, int* new_img) {
    int* d_data;
    int* d_hist;
    float* d_hist_p;
    int* zeros = new int[INTENSITIES];
    int length = rows * cols;
    int size = sizeof(int) * length;

    for (int i = 0; i < INTENSITIES; i++)
        zeros[i] = 0.0;

    // Allocate space for device 
    hipMalloc((void**)&d_data, size);
    hipMalloc((void**)&d_hist, sizeof(int) * INTENSITIES);
    hipMalloc((void**)&d_hist_p, sizeof(float) * INTENSITIES);

    // Copy inputs to device
    hipMemcpy(d_data, data, size, hipMemcpyHostToDevice);
    hipMemcpy(d_hist, zeros, INTENSITIES, hipMemcpyHostToDevice);
    hipMemcpy(d_hist_p, zeros, INTENSITIES, hipMemcpyHostToDevice);

    // Launch add() kernel on GPU
    int block_size = rows;
    int grid_size = ((length + block_size) / block_size);

    hist_frecuencies << < grid_size, block_size >> > (d_data, d_hist, d_hist_p, length);

    float* hist_p = new float[INTENSITIES];
    float* CH = new float[INTENSITIES];

    hipMemcpy(o_hist, d_hist, sizeof(int) * INTENSITIES, hipMemcpyDeviceToHost);
    hipMemcpy(hist_p, d_hist_p, sizeof(float) * INTENSITIES, hipMemcpyDeviceToHost);
    
    sum_probabilities(hist_p, CH, INTENSITIES);
    /*
    float acc = 0.0;
    for (int i = 0; i < 256; i++) {
        cout << CH[i] << ", ";
        acc += hist_p[i];
    }   
    cout << "acc: " << acc << endl << endl;
    */

    /// ////////////////////////////////////////////////////////////////
    /// rebuilt image
    /// ////////////////////////////////////////////////////////////////
    int* d_old_image, *d_new_image;
    float* d_CH;
    hipMalloc((void**)&d_old_image, sizeof(int) * length);
    hipMalloc((void**)&d_new_image, sizeof(int) * length);
    hipMalloc((void**)&d_CH, sizeof(float) * INTENSITIES);

    hipMemcpy(d_old_image, data, sizeof(int) * length, hipMemcpyHostToDevice);
    hipMemcpy(d_CH, CH, sizeof(float) * INTENSITIES, hipMemcpyHostToDevice);

    rebuild_img << < grid_size, block_size >> > (d_CH, d_new_image, d_old_image, length);    
    hipMemcpy(new_img, d_new_image, sizeof(int) * length, hipMemcpyDeviceToHost);

    //for (int i = 0; i < length; i++) {
    //    cout << new_img[i] << ", ";
    //}


    hipFree(d_data);
    hipFree(hist_p);
    hipFree(d_old_image);
    hipFree(d_new_image);
    hipFree(d_CH);
}




