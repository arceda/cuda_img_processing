#include "hip/hip_runtime.h"

#include "2_equalize.h"

#define INTENSITIES 256

__global__ void arithmetics(int* d_img_1, int* d_img_2, 
                            int* d_output_1,  int* d_output_2, int* d_output_3, int* d_output_4,
                            int len) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < len) {
        d_output_1[tid] = int(0.5*d_img_1[tid] + 0.5*d_img_2[tid]);
        d_output_2[tid] = d_img_1[tid] - d_img_2[tid];
        d_output_3[tid] = d_img_1[tid] * d_img_2[tid];
        d_output_4[tid] = d_img_1[tid] / d_img_2[tid];
    }
}

void arithmetics_cuda(  int* img_1, int* img_2, int rows, int cols, 
                        int* output_1, int* output_2, int* output_3, int* output_4) {
    int* d_img_1, * d_img_2;
    int* d_output_1, * d_output_2, * d_output_3, * d_output_4;
    int length = rows * cols;
    int size = sizeof(int) * length * 3;

    hipMalloc((void**)&d_img_1, size);
    hipMalloc((void**)&d_img_2, size);    
    hipMalloc((void**)&d_output_1, size);
    hipMalloc((void**)&d_output_2, size);
    hipMalloc((void**)&d_output_3, size);
    hipMalloc((void**)&d_output_4, size);

    hipMemcpy(d_img_1, img_1, size, hipMemcpyHostToDevice);
    hipMemcpy(d_img_2, img_2, size, hipMemcpyHostToDevice);

    int block_size = rows;
    int grid_size = ((length + block_size) / block_size);
    arithmetics << < grid_size, block_size >> > (   d_img_1, d_img_2, d_output_1, 
                                                    d_output_2, d_output_3, d_output_4, length);

    hipMemcpy(output_1,    d_output_1, size, hipMemcpyDeviceToHost);
    hipMemcpy(output_2,    d_output_2, size, hipMemcpyDeviceToHost);
    hipMemcpy(output_3,    d_output_3, size, hipMemcpyDeviceToHost);
    hipMemcpy(output_4,    d_output_4, size, hipMemcpyDeviceToHost);

    hipFree(d_img_1);
    hipFree(d_img_2);
    hipFree(d_output_1);
    hipFree(d_output_2);
    hipFree(d_output_3);
    hipFree(d_output_4);
}
