﻿
#include "1_histogram.h"
#include "2_equalize.h"
#include "3_global.h"
#include "4_arithmetics.h"
#include "5_conv.h"
#include "utils.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <iostream>

#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc.hpp>


using namespace cv;
using namespace std;

void get_histogram(Mat img) {
    imshow("Image", img);
    waitKey(0);
    int vector_size = img.rows * img.cols;
    //cout << "rows: " << img.rows << " cols: " << img.cols << endl;

    int* blue = new int[vector_size];
    int* green = new int[vector_size];
    int* red = new int[vector_size];

    int* output_blue = new int[256];
    int* output_green = new int[256];
    int* output_red = new int[256];

    mat_to_vec(img, blue, green, red);
    histogram_cuda(blue, green, red, img.rows, img.cols, output_blue, output_green, output_red);
    histDisplay(output_blue, "hist blue");
    histDisplay(output_green, "hist green");
    histDisplay(output_red, "hist red");

    /*show_vec(output[0], 256);
    show_vec(output[1], 256);
    show_vec(output[2], 256);

    int acc = 0;
    for (int i = 0; i < 256; i++)
        acc += output[0][i];
    cout << "total: " << acc;
    */
}

void equalize_hist(Mat img) {
    Mat grey;
    cvtColor(img, grey, COLOR_BGR2GRAY);
    int vector_size = grey.rows * grey.cols;
    int* data = new int[vector_size];
    //float* data_out = new float[256];
    int* hist = new int[256];
    int* new_img_vec = new int[vector_size];

       
    imshow("Original image", grey);
    waitKey(0);

    mat_to_vec(grey, data);
    equalize_hist_cuda(data, grey.rows, grey.cols, hist, new_img_vec);
    
    //histDisplay(hist, "hist");
    Mat new_img = vec_to_mat(new_img_vec, grey.rows, grey.cols);
    imshow("Image equalized", new_img);
    waitKey(0);
 
    //show_vec(new_img_vec, 256);
}

void aply_function(Mat img) {
    //int*** data = create_tensor_int(img.rows, img.cols, img.channels());
    //mat_to_tensor(img, data);
    //show_tensor(data, img.rows, img.cols, img.channels());

    imshow("Image", img);
    waitKey(0);
    int vector_size = img.rows * img.cols;
    //cout << "rows: " << img.rows << " cols: " << img.cols << endl;

    int* blue = new int[vector_size];
    int* green = new int[vector_size];
    int* red = new int[vector_size];

    int* output_blue = new int[vector_size];
    int* output_green = new int[vector_size];
    int* output_red = new int[vector_size];

    mat_to_vec(img, blue, green, red);
    global_func_cuda(blue, green, red, img.rows, img.cols, output_blue, output_green, output_red);
    show_vec(output_blue, 100);

    Mat new_img = vec_to_mat(output_blue, output_green, output_red, img.rows, img.cols);
    imshow("Image global", new_img);
    waitKey(0);
}

void arithmetics(Mat img_1, Mat img_2) {      
    int vector_size = img_1.rows * img_1.cols;
    //cout << "rows: " << img.rows << " cols: " << img.cols << endl;

    int* img_vec_1 = new int[vector_size * 3];
    int* img_vec_2 = new int[vector_size * 3];
    int* output_1 = new int[vector_size * 3];
    int* output_2 = new int[vector_size * 3];
    int* output_3 = new int[vector_size * 3];
    int* output_4 = new int[vector_size * 3];

    mat_to_vec_1d(img_1, img_vec_1);
    mat_to_vec_1d(img_2, img_vec_2);
    //show_vec(img_vec_1, 100);      
    
    arithmetics_cuda(img_vec_1, img_vec_2, img_1.rows, img_1.cols, output_1, output_2, output_3, output_4);
    //show_vec(output_1, 100);
    Mat img_sum = vec_1d_to_mat(output_1, img_1.rows, img_1.cols);
    Mat img_mul = vec_1d_to_mat(output_2, img_1.rows, img_1.cols);
    Mat img_sub = vec_1d_to_mat(output_3, img_1.rows, img_1.cols);
    Mat img_div = vec_1d_to_mat(output_4, img_1.rows, img_1.cols);

    imshow("Image 1", img_1);    waitKey(0);
    imshow("Image 2", img_2);    waitKey(0);    
    imshow("sum", img_sum);    waitKey(0);      
    imshow("mul", img_mul);    waitKey(0);
    imshow("sub", img_sub);    waitKey(0);
    imshow("div", img_div);    waitKey(0);
}

void convolutions(Mat img, float *kernel, int kernel_rows, int kernel_cols) {
    int vector_size = img.rows * img.cols;
    int* img_vec = new int[vector_size * 3];    
    int* output = new int[vector_size * 3];
    
    mat_to_vec_1d(img, img_vec);         
    conv_cuda(img_vec, kernel, img.rows, img.cols, img.channels(), kernel_rows, kernel_cols, output);
    Mat img_result = vec_1d_to_mat(output, img.rows, img.cols);    

    imshow("Image", img);               waitKey(0);
    imshow("New Image", img_result);    waitKey(0);
}

int main() {    
    // PREGUNTA 1
    Mat img = imread("D:\\CUDA\\HelloCUDAopenCV\\lena.jpg");
    //get_histogram(img);    

    // PREGUNTA 2
    img = imread("D:\\CUDA\\HelloCUDAopenCV\\lena2.jpg");
    //equalize_hist(img);

    // PREGUNTA 3
    img = imread("D:\\CUDA\\HelloCUDAopenCV\\lena.jpg");
    //aply_function(img); 

    // PREGUNTA 4
    Mat img_1 = imread("D:\\CUDA\\HelloCUDAopenCV\\leon.jpg");
    Mat img_2 = imread("D:\\CUDA\\HelloCUDAopenCV\\aqp.jpg");
    Mat img_3 = imread("D:\\CUDA\\HelloCUDAopenCV\\sub_10.jpg");
    Mat img_4 = imread("D:\\CUDA\\HelloCUDAopenCV\\sub_11.jpg");
    //arithmetics(img_1, img_2);
    //arithmetics(img_3, img_4);

    // PREGUNTA 5
    img = imread("D:\\CUDA\\HelloCUDAopenCV\\lena.jpg");
    float kernel[] = {
            0.04, 0.04, 0.04, 0.04, 0.04,
            0.04, 0.04, 0.04, 0.04, 0.04,
            0.04, 0.04, 0.04, 0.04, 0.04,
            0.04, 0.04, 0.04, 0.04, 0.04,
            0.04, 0.04, 0.04, 0.04, 0.04
    }; // mean
    convolutions(img, kernel, 5, 5);

    img = imread("D:\\CUDA\\HelloCUDAopenCV\\sub_10.jpg");
    float kernel_sobel[] = {
            2, 1, 0, -1, -2,
            2, 1, 0, -1, -2,
            4, 2, 0, -2, -4,
            2, 1, 0, -1, -2,
            2, 1, 0, -1, -2,
    };
    convolutions(img, kernel_sobel, 5, 5);
    return 0;

}


