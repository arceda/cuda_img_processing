﻿
#include "1_histogram.h"
#include "2_equalize.h"
#include "3_global.h"
#include "4_arithmetics.h"
#include "5_conv.h"
#include "6_zoon.h"
#include "7_geometrics.h"
#include "utils.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <iostream>

#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc.hpp>


using namespace cv;
using namespace std;

void get_histogram(Mat img) {
    imshow("Image", img);
    waitKey(0);
    int vector_size = img.rows * img.cols;
    //cout << "rows: " << img.rows << " cols: " << img.cols << endl;

    int* blue = new int[vector_size];
    int* green = new int[vector_size];
    int* red = new int[vector_size];

    int* output_blue = new int[256];
    int* output_green = new int[256];
    int* output_red = new int[256];

    mat_to_vec(img, blue, green, red);
    histogram_cuda(blue, green, red, img.rows, img.cols, output_blue, output_green, output_red);
    histDisplay(output_blue, "hist blue");
    histDisplay(output_green, "hist green");
    histDisplay(output_red, "hist red");

    /*show_vec(output[0], 256);
    show_vec(output[1], 256);
    show_vec(output[2], 256);

    int acc = 0;
    for (int i = 0; i < 256; i++)
        acc += output[0][i];
    cout << "total: " << acc;
    */
}

void equalize_hist(Mat img) {
    Mat grey;
    cvtColor(img, grey, COLOR_BGR2GRAY);
    int vector_size = grey.rows * grey.cols;
    int* data = new int[vector_size];
    //float* data_out = new float[256];
    int* hist = new int[256];
    int* new_img_vec = new int[vector_size];

       
    imshow("Original image", grey);
    waitKey(0);

    mat_to_vec(grey, data);
    equalize_hist_cuda(data, grey.rows, grey.cols, hist, new_img_vec);
    
    //histDisplay(hist, "hist");
    Mat new_img = vec_to_mat(new_img_vec, grey.rows, grey.cols);
    imshow("Image equalized", new_img);
    waitKey(0);
 
    //show_vec(new_img_vec, 256);
}

void aply_function(Mat img) {
    //int*** data = create_tensor_int(img.rows, img.cols, img.channels());
    //mat_to_tensor(img, data);
    //show_tensor(data, img.rows, img.cols, img.channels());

    imshow("Image", img);
    waitKey(0);
    int vector_size = img.rows * img.cols;
    //cout << "rows: " << img.rows << " cols: " << img.cols << endl;

    int* blue = new int[vector_size];
    int* green = new int[vector_size];
    int* red = new int[vector_size];

    int* output_blue = new int[vector_size];
    int* output_green = new int[vector_size];
    int* output_red = new int[vector_size];

    mat_to_vec(img, blue, green, red);
    global_func_cuda(blue, green, red, img.rows, img.cols, output_blue, output_green, output_red);
    show_vec(output_blue, 100);

    Mat new_img = vec_to_mat(output_blue, output_green, output_red, img.rows, img.cols);
    imshow("Image global", new_img);
    waitKey(0);
}

void arithmetics(Mat img_1, Mat img_2) {      
    int vector_size = img_1.rows * img_1.cols;
    //cout << "rows: " << img.rows << " cols: " << img.cols << endl;

    int* img_vec_1 = new int[vector_size * 3];
    int* img_vec_2 = new int[vector_size * 3];
    int* output_1 = new int[vector_size * 3];
    int* output_2 = new int[vector_size * 3];
    int* output_3 = new int[vector_size * 3];
    int* output_4 = new int[vector_size * 3];

    mat_to_vec_1d(img_1, img_vec_1);
    mat_to_vec_1d(img_2, img_vec_2);
    //show_vec(img_vec_1, 100);      
    
    arithmetics_cuda(img_vec_1, img_vec_2, img_1.rows, img_1.cols, output_1, output_2, output_3, output_4);
    //show_vec(output_1, 100);
    Mat img_sum = vec_1d_to_mat(output_1, img_1.rows, img_1.cols);
    Mat img_mul = vec_1d_to_mat(output_2, img_1.rows, img_1.cols);
    Mat img_sub = vec_1d_to_mat(output_3, img_1.rows, img_1.cols);
    Mat img_div = vec_1d_to_mat(output_4, img_1.rows, img_1.cols);

    imshow("Image 1", img_1);    waitKey(0);
    imshow("Image 2", img_2);    waitKey(0);    
    imshow("sum", img_sum);    waitKey(0);      
    imshow("mul", img_mul);    waitKey(0);
    imshow("sub", img_sub);    waitKey(0);
    imshow("div", img_div);    waitKey(0);
}

void convolutions(Mat img, float *kernel, int kernel_rows, int kernel_cols) {
    int vector_size = img.rows * img.cols;
    int* img_vec = new int[vector_size * 3];    
    int* output = new int[vector_size * 3];
    
    mat_to_vec_1d(img, img_vec);         
    conv_cuda(img_vec, kernel, img.rows, img.cols, img.channels(), kernel_rows, kernel_cols, output);
    Mat img_result = vec_1d_to_mat(output, img.rows, img.cols);    

    imshow("Image", img);               waitKey(0);
    imshow("New Image", img_result);    waitKey(0);
}

void zoon(Mat img) {
    Mat grey;
    cvtColor(img, grey, COLOR_BGR2GRAY);
    int vector_size = img.rows * img.cols;
    //int* img_vec = new int[vector_size * img.channels()];
    //int* output = new int[vector_size * 2 * img.channels()];
    //mat_to_vec_1d(img, img_vec);
    //zoon_cuda(img_vec, img.rows, img.cols, img.channels(),  output);
    //Mat img_result = vec_1d_to_mat(output, img.rows, img.cols);


    /*int length = 9;
    int rows = 3; int cols = 3;
    int* img_vec = new int[length];
    img_vec[0] = 100; img_vec[1] = 200; img_vec[2] = 300; img_vec[3] = 400; img_vec[4] = 500;
    img_vec[5] = 600; img_vec[6] = 700; img_vec[7] = 800; img_vec[8] = 900;
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            cout << img_vec[i * cols + j] << " ";
        }
        cout << endl;
    }
    cout << endl;

    int* output = new int[length * 4];
    zoon_cuda(img_vec, rows, cols, 1, output);
    
    cout << endl << "output\n";
    for (int i = 0; i < rows*2; i++) {
        for (int j = 0; j < cols*2; j++) {
            cout << output[i * cols * 2 + j] << " ";
        }
        cout << endl;
    }
    cout << endl;
    */



    int* img_vec = new int[vector_size];
    int* output = new int[vector_size * 4];
    mat_to_vec(grey, img_vec);
    zoon_cuda(img_vec, grey.rows, grey.cols, grey.channels(),  output);
    Mat img_result = vec_to_mat(output, grey.rows*2, grey.cols*2);

    show_vec(img_vec, 20);
    cout << endl;
    show_vec(output, 400);

    imshow("Image low resolution", grey);                    waitKey(0);
    imshow("Image high resolution", img_result);            waitKey(0);
    
}

void geometrics(Mat img, int* x_1, int* y_1, int* x_2, int* y_2) {
    float* M = new float[6]; // coeficient of M matrix for affine transformation
    AffineSolver(M, x_1, y_1, x_2, y_2);
    PrintMatrix(M);    

    Mat img_input = img;
    //cvtColor(img, img_input, COLOR_BGR2GRAY);
    int vector_size = img_input.rows * img_input.cols * img_input.channels();

    int* img_vec = new int[ vector_size ];
    int* output = new int[ vector_size ];
    mat_to_vec_1d(img_input, img_vec);
    geometrics_cuda(img_vec, M, img_input.rows, img_input.cols, img_input.channels(), output);
    Mat img_result = vec_1d_to_mat(output, img_input.rows, img_input.cols);

    imshow("Original", img_input);    waitKey(0);
    imshow("Affine", img_result);     waitKey(0);

}

int main() {    
    // PREGUNTA 1
    Mat img = imread("D:\\CUDA\\HelloCUDAopenCV\\lena.jpg");
    //get_histogram(img);    

    // PREGUNTA 2
    img = imread("D:\\CUDA\\HelloCUDAopenCV\\lena2.jpg");
    //equalize_hist(img);

    // PREGUNTA 3
    img = imread("D:\\CUDA\\HelloCUDAopenCV\\lena.jpg");
    //aply_function(img); 

    // PREGUNTA 4
    Mat img_1 = imread("D:\\CUDA\\HelloCUDAopenCV\\leon.jpg");
    Mat img_2 = imread("D:\\CUDA\\HelloCUDAopenCV\\aqp.jpg");
    Mat img_3 = imread("D:\\CUDA\\HelloCUDAopenCV\\sub_10.jpg");
    Mat img_4 = imread("D:\\CUDA\\HelloCUDAopenCV\\sub_11.jpg");
    //arithmetics(img_1, img_2);
    //arithmetics(img_3, img_4);

    // PREGUNTA 5
    img = imread("D:\\CUDA\\HelloCUDAopenCV\\lena.jpg");
    float kernel[] = {
            0.04, 0.04, 0.04, 0.04, 0.04,
            0.04, 0.04, 0.04, 0.04, 0.04,
            0.04, 0.04, 0.04, 0.04, 0.04,
            0.04, 0.04, 0.04, 0.04, 0.04,
            0.04, 0.04, 0.04, 0.04, 0.04
    }; // mean
    //convolutions(img, kernel, 5, 5);

    img = imread("D:\\CUDA\\HelloCUDAopenCV\\sub_10.jpg");
    float kernel_sobel[] = {
            2, 1, 0, -1, -2,
            2, 1, 0, -1, -2,
            4, 2, 0, -2, -4,
            2, 1, 0, -1, -2,
            2, 1, 0, -1, -2,
    };
    //convolutions(img, kernel_sobel, 5, 5);
    

    // PREGUNTA 6
    img = imread("D:\\CUDA\\HelloCUDAopenCV\\orange.jpg");
    //zoon(img);

    // PREGUNTA 7
    img = imread("D:\\CUDA\\HelloCUDAopenCV\\orange.jpg");
    int x_1[] = { 0, 0, 10 };   int y_1[] = { 10, 0, 0 };    int x_2[] = { 0, 0, 5 };    int y_2[] = { 5, 0, 0 };
    int x_3[] = { 50, 200, 50 };   int y_3[] = { 50, 50, 200 };    int x_4[] = { 10, 200, 100 };    int y_4[] = { 100, 50, 250 };    
    geometrics(img, x_1, y_1, x_2, y_2);
    geometrics(img, x_3, y_3, x_4, y_4);

    return 0;

}


